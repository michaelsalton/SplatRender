#include "hip/hip_runtime.h"
#include "cuda_rasterizer.h"
#include "cuda_rasterizer_wrapper.h"
#include "cuda_manager.h"
#include "cuda_constants.h"
#include "kernels/kernels.h"
#include "kernel_profiler.h"
#include "../core/camera.h"
#include <iostream>
#include <algorithm>
#include <glm/gtc/type_ptr.hpp>

namespace SplatRender {
namespace CUDA {

// Helper function to convert Gaussian3D to GaussianData3D
void convertGaussians(const std::vector<Gaussian3D>& gaussians, GaussianData3D* output) {
    for (size_t i = 0; i < gaussians.size(); i++) {
        const Gaussian3D& src = gaussians[i];
        GaussianData3D& dst = output[i];
        
        dst.position = make_float3(src.position.x, src.position.y, src.position.z);
        dst.opacity = src.opacity;
        dst.scale = make_float3(src.scale.x, src.scale.y, src.scale.z);
        dst.rotation = make_float4(src.rotation.x, src.rotation.y, src.rotation.z, src.rotation.w);
        
        // Copy spherical harmonics coefficients
        for (int j = 0; j < 45; j++) {
            dst.sh_coeffs[j] = src.sh_coeffs[j];
        }
        // Zero out unused coefficients
        for (int j = 45; j < 48; j++) {
            dst.sh_coeffs[j] = 0.0f;
        }
    }
}

CudaRasterizer::CudaRasterizer() 
    : initialized_(false), max_gaussians_(0) {
    timer_ = std::make_unique<CudaTimer>();
}

CudaRasterizer::~CudaRasterizer() {
    shutdown();
}

bool CudaRasterizer::initialize(const RenderSettings& settings) {
    if (initialized_) {
        return true;
    }
    
    // Check if CUDA is available
    if (!CudaManager::getInstance().isInitialized()) {
        if (!CudaManager::getInstance().initialize()) {
            std::cerr << "Failed to initialize CUDA" << std::endl;
            return false;
        }
    }
    
    settings_ = settings;
    
    // Allocate initial buffers (will grow as needed)
    size_t initial_capacity = 10000;
    allocateBuffers(initial_capacity);
    
    // Create pinned memory for output
    size_t output_size = settings.width * settings.height * 4; // RGBA
    h_pinned_output_ = std::make_unique<PinnedMemory<float>>(output_size);
    
    // Initialize OpenGL interop
    gl_interop_ = std::make_unique<CudaGLInterop>();
    
    initialized_ = true;
    
    std::cout << "CUDA Rasterizer initialized successfully" << std::endl;
    std::cout << "  Resolution: " << settings.width << "x" << settings.height << std::endl;
    std::cout << "  Tile Size: " << settings.tile_size << "x" << settings.tile_size << std::endl;
    std::cout << "  Initial Capacity: " << initial_capacity << " Gaussians" << std::endl;
    
    return true;
}

void CudaRasterizer::shutdown() {
    if (!initialized_) {
        return;
    }
    
    freeBuffers();
    h_pinned_output_.reset();
    gl_interop_.reset();
    
    initialized_ = false;
}

void CudaRasterizer::render(const std::vector<Gaussian3D>& gaussians,
                            const Camera& camera,
                            std::vector<float>& output_buffer) {
    if (!initialized_) {
        std::cerr << "CUDA Rasterizer not initialized" << std::endl;
        return;
    }
    
    if (gaussians.empty()) {
        // Clear output buffer
        size_t buffer_size = settings_.width * settings_.height * 4;
        output_buffer.resize(buffer_size);
        std::fill(output_buffer.begin(), output_buffer.end(), 0.0f);
        return;
    }
    
    // Ensure buffers are large enough
    ensureBufferSizes(gaussians.size());
    
    // Get CUDA stream
    hipStream_t stream = CudaManager::getInstance().getDefaultStream();
    
    // Declare variables that need to persist across scopes
    CudaMemory<GaussianData3D> d_gaussians_3d_data(gaussians.size());
    CudaMemory<GaussianData2D> d_gaussians_2d_data(gaussians.size());
    CudaMemory<int> d_visible_count(1);
    int h_visible_count;
    
    // Get render params
    float aspect_ratio = static_cast<float>(settings_.width) / static_cast<float>(settings_.height);
    glm::mat4 view_matrix = camera.getViewMatrix();
    glm::mat4 proj_matrix = camera.getProjectionMatrix(aspect_ratio);
    glm::vec3 cam_pos = camera.getPosition();
    
    RenderParams render_params = createRenderParams(
        settings_.width,
        settings_.height,
        gaussians.size()
    );
    int total_tiles = render_params.total_tiles;
    
    // Tile buffers
    CudaMemory<int> d_tile_lists(total_tiles * MAX_GAUSSIANS_PER_TILE);
    CudaMemory<int> d_tile_counts(total_tiles);
    CudaMemory<float> d_tile_depths(total_tiles * MAX_GAUSSIANS_PER_TILE);
    CudaMemory<int> d_tile_offsets(total_tiles);
    CudaMemory<int> d_tile_lists_compact(gaussians.size() * 4);  // Will be resized after projection
    CudaMemory<float> d_tile_depths_compact(gaussians.size() * 4);
    CudaMemory<float4> d_output_image(settings_.width * settings_.height);
    
    // ========================================================================
    // Step 1: Upload Gaussians to GPU
    // ========================================================================
    {
        splat::KernelProfiler::ScopedTimer upload_timer("Upload");
        
        // Allocate pinned memory for conversion
        PinnedMemory<GaussianData3D> h_gaussians_3d(gaussians.size());
        convertGaussians(gaussians, h_gaussians_3d.getHostPtr());
        
        // Upload to device
        d_gaussians_3d_data.copyFromHostAsync(h_gaussians_3d.getHostPtr(), gaussians.size(), stream);
        
        // Record memory allocation
        splat::MemoryProfiler::getInstance().recordAllocation("gaussians_3d", 
            d_gaussians_3d_data.allocatedBytes());
    }
    stats_.upload_time_ms = splat::KernelProfiler::getInstance().getLastTime("Upload");
    
    // ========================================================================
    // Step 2: Projection Kernel
    // ========================================================================
    {  
        splat::KernelProfiler::ScopedTimer proj_timer("Projection");
    
    // Create camera parameters
    float cam_pos_array[3] = {cam_pos.x, cam_pos.y, cam_pos.z};
    
    CameraParams camera_params = createCameraParams(
        glm::value_ptr(view_matrix),
        glm::value_ptr(proj_matrix),
        cam_pos_array,
        camera.getFOV() * M_PI / 180.0f,  // Convert to radians
        camera.getFOV() * settings_.height / settings_.width * M_PI / 180.0f,
        settings_.width,
        settings_.height
    );
    
    // Launch projection kernel
    launchProjectionKernel(
        d_gaussians_3d_data.getDevicePtr(),
        d_gaussians_2d_data.getDevicePtr(),
        d_visible_count.getDevicePtr(),
        camera_params,
        render_params,
        gaussians.size(),
        stream
    );
    
    // Get visible count
    d_visible_count.copyToHost(&h_visible_count, 1);
    stats_.visible_gaussians = h_visible_count;
    
    }
    stats_.projection_time_ms = splat::KernelProfiler::getInstance().getLastTime("Projection");
    
    // ========================================================================
    // Step 3: Tiling Kernel
    // ========================================================================
    {
        splat::KernelProfiler::ScopedTimer tiling_timer("Tiling");
    
    // Launch tiling kernel
    launchTilingKernel(
        d_gaussians_2d_data.getDevicePtr(),
        d_tile_lists.getDevicePtr(),
        d_tile_counts.getDevicePtr(),
        d_tile_depths.getDevicePtr(),
        h_visible_count,
        render_params,
        stream
    );
    
    // Compact tile lists
    
    launchCompactionKernel(
        d_tile_lists.getDevicePtr(),
        d_tile_depths.getDevicePtr(),
        d_tile_lists_compact.getDevicePtr(),
        d_tile_depths_compact.getDevicePtr(),
        d_tile_offsets.getDevicePtr(),
        d_tile_counts.getDevicePtr(),
        total_tiles,
        stream
    );
    
    }
    float tiling_time = splat::KernelProfiler::getInstance().getLastTime("Tiling");
    
    // ========================================================================
    // Step 4: Sorting Kernel
    // ========================================================================
    {
        splat::KernelProfiler::ScopedTimer sorting_timer("Sorting");
    
    launchSortingKernel(
        d_tile_lists_compact.getDevicePtr(),
        d_tile_depths_compact.getDevicePtr(),
        d_tile_counts.getDevicePtr(),
        d_tile_offsets.getDevicePtr(),
        total_tiles,
        stream
    );
    
    }
    stats_.sorting_time_ms = splat::KernelProfiler::getInstance().getLastTime("Sorting") + tiling_time;
    
    // ========================================================================
    // Step 5: Rasterization Kernel
    // ========================================================================
    {
        splat::KernelProfiler::ScopedTimer raster_timer("Rasterization");
    
    // Clear image
    launchClearImageKernel(
        d_output_image.getDevicePtr(),
        settings_.width,
        settings_.height,
        make_float4(0.0f, 0.0f, 0.0f, 0.0f),
        stream
    );
    
    // Launch rasterization kernel
    launchRasterizationKernel(
        d_gaussians_2d_data.getDevicePtr(),
        d_tile_lists_compact.getDevicePtr(),
        d_tile_counts.getDevicePtr(),
        d_tile_offsets.getDevicePtr(),
        d_output_image.getDevicePtr(),
        render_params,
        stream,
        false  // Use tile-based version
    );
    
    }
    stats_.rasterization_time_ms = splat::KernelProfiler::getInstance().getLastTime("Rasterization");
    
    // ========================================================================
    // Step 6: Download result to CPU
    // ========================================================================
    {
        splat::KernelProfiler::ScopedTimer download_timer("Download");
    
    // Synchronize to ensure kernels are complete
    hipStreamSynchronize(stream);
    
    // Resize output buffer
    size_t output_size = settings_.width * settings_.height * 4;
    output_buffer.resize(output_size);
    
    // Copy from device (float4) to host (float array)
    d_output_image.copyToHost(reinterpret_cast<float4*>(output_buffer.data()), 
                              settings_.width * settings_.height);
    
    }
    stats_.download_time_ms = splat::KernelProfiler::getInstance().getLastTime("Download");
    
    // Update total time
    stats_.total_time_ms = stats_.upload_time_ms + stats_.projection_time_ms + 
                           stats_.sorting_time_ms + stats_.rasterization_time_ms + 
                           stats_.download_time_ms;
    
    // Update memory usage
    stats_.memory_used_bytes = d_gaussians_3d_data.allocatedBytes() + 
                               d_gaussians_2d_data.allocatedBytes() +
                               d_output_image.allocatedBytes() +
                               d_tile_lists.allocatedBytes() +
                               d_tile_counts.allocatedBytes();
    
    // Update performance monitor
    auto& perf_monitor = splat::PerformanceMonitor::getInstance();
    perf_monitor.updateGaussianStats(stats_.visible_gaussians, 
                                     gaussians.size() - stats_.visible_gaussians);
    
    // Update frame stats for profiler
    splat::KernelProfiler::FrameStats frame_stats;
    frame_stats.projection_ms = stats_.projection_time_ms;
    frame_stats.tiling_ms = tiling_time;
    frame_stats.sorting_ms = stats_.sorting_time_ms - tiling_time;
    frame_stats.rasterization_ms = stats_.rasterization_time_ms;
    frame_stats.total_ms = stats_.total_time_ms;
    frame_stats.rendered_gaussians = stats_.visible_gaussians;
    frame_stats.culled_gaussians = gaussians.size() - stats_.visible_gaussians;
    splat::KernelProfiler::getInstance().updateFrameStats(frame_stats);
}

void CudaRasterizer::renderDirect(const Gaussian3D* d_gaussians, 
                                  size_t count,
                                  const Camera& camera,
                                  hipSurfaceObject_t surface) {
    if (!initialized_) {
        return;
    }
    
    // This will be implemented with actual kernels in Phase 8
    // For now, it's a placeholder
}

void CudaRasterizer::renderToTexture(const std::vector<Gaussian3D>& gaussians,
                                     const Camera& camera,
                                     GLuint texture) {
    if (!initialized_ || !gl_interop_) {
        return;
    }
    
    // Register texture if not already registered
    if (!gl_interop_->isTextureRegistered()) {
        gl_interop_->registerTexture(texture, settings_.width, settings_.height);
    }
    
    // Upload Gaussians
    uploadGaussians(gaussians);
    
    // Map texture for CUDA
    hipSurfaceObject_t surface = gl_interop_->mapTextureForCuda();
    
    // Render directly to surface
    renderDirect(d_gaussians_3d_.getDevicePtr(), gaussians.size(), camera, surface);
    
    // Unmap texture
    gl_interop_->unmapTexture();
}

void CudaRasterizer::uploadGaussians(const std::vector<Gaussian3D>& gaussians) {
    ensureBufferSizes(gaussians.size());
    d_gaussians_3d_.copyFromHost(gaussians.data(), gaussians.size());
    stats_.visible_gaussians = static_cast<int>(gaussians.size());
}

void CudaRasterizer::allocateBuffers(size_t max_gaussians) {
    max_gaussians_ = max_gaussians;
    
    // Allocate device buffers
    d_gaussians_3d_.allocate(max_gaussians);
    d_gaussians_2d_.allocate(max_gaussians);
    d_visible_indices_.allocate(max_gaussians);
    
    // Allocate output and depth buffers
    size_t pixel_count = settings_.width * settings_.height;
    d_output_buffer_.allocate(pixel_count * 4); // RGBA
    d_depth_buffer_.allocate(pixel_count);
    
    // Allocate tile buffers
    int tile_count = getTileCount();
    d_tile_lists_.allocate(tile_count * max_gaussians); // Worst case
    d_tile_counts_.allocate(tile_count);
}

void CudaRasterizer::freeBuffers() {
    d_gaussians_3d_.free();
    d_gaussians_2d_.free();
    d_output_buffer_.free();
    d_depth_buffer_.free();
    d_tile_lists_.free();
    d_tile_counts_.free();
    d_visible_indices_.free();
}

void CudaRasterizer::setSettings(const RenderSettings& settings) {
    if (settings.width != settings_.width || settings.height != settings_.height) {
        // Need to reallocate output buffers
        settings_ = settings;
        
        size_t pixel_count = settings_.width * settings_.height;
        d_output_buffer_.allocate(pixel_count * 4);
        d_depth_buffer_.allocate(pixel_count);
        
        int tile_count = getTileCount();
        d_tile_lists_.allocate(tile_count * max_gaussians_);
        d_tile_counts_.allocate(tile_count);
        
        // Reallocate pinned memory
        h_pinned_output_ = std::make_unique<PinnedMemory<float>>(pixel_count * 4);
    } else {
        settings_ = settings;
    }
}

void CudaRasterizer::ensureBufferSizes(size_t num_gaussians) {
    if (num_gaussians > max_gaussians_) {
        // Grow buffers by 50% to avoid frequent reallocations
        size_t new_capacity = static_cast<size_t>(num_gaussians * 1.5);
        allocateBuffers(new_capacity);
    }
}

int CudaRasterizer::getTileCount() const {
    int tiles_x = (settings_.width + settings_.tile_size - 1) / settings_.tile_size;
    int tiles_y = (settings_.height + settings_.tile_size - 1) / settings_.tile_size;
    return tiles_x * tiles_y;
}

// Factory function
std::unique_ptr<CPURasterizer> createRasterizer(bool prefer_cuda) {
    if (prefer_cuda && isCudaAvailable()) {
        try {
            auto wrapper = std::make_unique<CudaRasterizerWrapper>();
            std::cout << "Using CUDA-accelerated rasterizer" << std::endl;
            return wrapper;
        } catch (const std::exception& e) {
            std::cerr << "Failed to create CUDA rasterizer: " << e.what() << std::endl;
            std::cerr << "Falling back to CPU rasterizer" << std::endl;
        }
    }
    return std::make_unique<CPURasterizer>();
}

} // namespace CUDA
} // namespace SplatRender